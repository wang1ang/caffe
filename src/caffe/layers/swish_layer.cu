#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/swish_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SwishForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (0.5 * tanh(0.5 * in[index]) + 0.5);
  }
}

template <typename Dtype>
void SwishLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SwishForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void SwishBackward(const int n, const Dtype* in_data, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype swish_x = out_data[index];
	const Dtype sigmoid_x = 0.5 * tanh(0.5 * in_data[index]) + 0.5;
    out_diff[index] = in_diff[index] * (sigmoid_x + swish_x * (1 - sigmoid_x));
  }
}

template <typename Dtype>
void SwishLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
	const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SwishBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_diff, top_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SwishLayer);


}  // namespace caffe
